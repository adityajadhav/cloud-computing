
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>

#define NUM_ITERATIONS 100000000000
#define NUM_THREADS 3072


__global__ void solve_equation_float(float * d_out, float * d_in) {

	int x = threadIdx.x;
	long i;
	for (i = 0; i < NUM_ITERATIONS; i++) {
		d_out[x] =  5.66 + 7.888 + 4.67 + 8.99
		            + 5.69 + 7.898 + 4.47 + 8.39
		            + 5.66 + 7.888 + 4.67 + 8.99
		            + 5.69 + 7.898 + 4.47 + 8.39
		            + 5.66 + 7.888 + 4.67 + 8.99
		            + 5.66 + 7.888 + 4.67 + 8.99
		            + 5.69 + 7.898 + 4.47 + 8.39
		            + 5.66 + 7.888 + 4.67 + 8.99
		            + 5.69 + 7.898 + 4.47 + 8.39
		            + 5.66 + 7.888 + 4.67 + 8.99;
	}
}

__global__ void solve_equation_int(int * d_out, int * d_in) {

	int x = threadIdx.x;
	long i;

	for (i = 0; i < NUM_ITERATIONS; i++) {
		d_out[x]  = 566 + 7888 + 467 + 899
		            + 569 + 7898 + 447 + 839
		            + 566 + 7888 + 467 + 899
		            + 569 + 7898 + 447 + 839
		            + 566 + 7888 + 467 + 899
		            + 569 + 7898 + 447 + 839
		            + 566 + 7888 + 467 + 899
		            + 569 + 7898 + 447 + 839
		            + 566 + 7888 + 467 + 899
		            + 569 + 7898 + 447;
	}
}


void calculateGIOPS() {

	clock_t start;
	FILE *logFile;
	double timeTaken;

	logFile = fopen("gpu-benchmarking.log", "a");

	const int ARRAY_SIZE = NUM_THREADS * sizeof(int);

	int h_in[NUM_THREADS];

	for (int i = 0; i < NUM_THREADS; i++) {
		h_in[i] = i;
	}

	int h_out[NUM_THREADS];

	int * d_in;
	int * d_out;


	start = clock();


	hipMalloc((void**) &d_in, ARRAY_SIZE);
	hipMalloc((void**) &d_out, ARRAY_SIZE);

	hipMemcpy(d_in, h_in, ARRAY_SIZE, hipMemcpyHostToDevice);


	// launch the kernel
	solve_equation_int <<< 1, NUM_THREADS>>>(d_out, d_in);

	hipDeviceSynchronize();

	start = clock() - start;


	hipMemcpy(h_out, d_out, ARRAY_SIZE, hipMemcpyDeviceToHost);

	hipFree(d_in);
	hipFree(d_out);

	timeTaken = (double) start / CLOCKS_PER_SEC;


	printf("time taken -> %lf s\n", timeTaken);
	double flops, gflops;

	flops = NUM_ITERATIONS * (40 / timeTaken);
	gflops = flops / 1000000000;

	fprintf(logFile, "\n\n\nGIOPS Result");
	fprintf(logFile, "\nThreads: %d", NUM_THREADS);
	fprintf(logFile, "\nCPU Time Used: %lf ms", timeTaken);
	fprintf(logFile, "\nNumber of IOPS: %lf", flops);
	fprintf(logFile, "\nNumber of GIOPS: %lf", gflops);


	printf("Please check output file -> gpu-benchmarking.log\n");
	printf("benchmark success\n");


}

void calculateGFLOPS() {

	clock_t start;
	FILE *logFile;
	double timeTaken = 0;

	logFile = fopen("gpu-benchmarking.log", "a");

	const int ARRAY_SIZE = NUM_THREADS * sizeof(float);

	float h_in[NUM_THREADS];

	for (int i = 0; i < NUM_THREADS; i++) {
		h_in[i] = float(i);
	}

	float h_out[NUM_THREADS];

	float * d_in;
	float * d_out;

	start = clock();

	hipMalloc((void**) &d_in, ARRAY_SIZE);
	hipMalloc((void**) &d_out, ARRAY_SIZE);

	hipMemcpy(d_in, h_in, ARRAY_SIZE, hipMemcpyHostToDevice);

	// launch the kernel
	solve_equation_float <<< 1, NUM_THREADS>>>(d_out, d_in);

	hipDeviceSynchronize();

	start = clock() - start;


	hipMemcpy(h_out, d_out, ARRAY_SIZE, hipMemcpyDeviceToHost);

	hipFree(d_in);
	hipFree(d_out);

	timeTaken = (double) start / CLOCKS_PER_SEC;

	printf("time taken -> %lf s\n", timeTaken);
	double flops, gflops;

	flops = NUM_ITERATIONS * (40 / timeTaken);
	gflops = flops / 1000000000;

	fprintf(logFile, "\n\n\nGFLOPS Result");
	fprintf(logFile, "\nThreads: %d", NUM_THREADS);
	fprintf(logFile, "\nCPU Time Used: %lf ms", timeTaken);
	fprintf(logFile, "\nNumber of FLOPS: %lf", flops);
	fprintf(logFile, "\nNumber of GFLOPS: %lf", gflops);

	fclose(logFile);
	printf("Please check output file -> gpu-benchmarking.log\n");
	printf("benchmark success\n");



}






int main(int argc, char ** argv) {

	int mode = 0;
	if (argc != 2) {
		printf("usage: ./a.out <mode>\n");
		printf("1 -> Calculate GIOPS\n");
		printf("2 -> Calculate GFLOPS\n");
		return EXIT_FAILURE;
	}

	mode = atoi(argv[1]);

	if (mode == 1)
		calculateGIOPS();
	else if (mode == 2)
		calculateGFLOPS();
	else
		printf("Unknown mode");

	return 0;
}
